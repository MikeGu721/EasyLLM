#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// SPDX-License-Identifier: Apache-2.0

// DeepSpeed Team

#include "inference_cuda_layers.h"

#ifndef __HIP_PLATFORM_HCC__
#include <hip/hip_runtime_api.h>
#endif

namespace cg = cooperative_groups;
namespace cg = cooperative_groups;

__global__ void apply_rotary_pos_emb(float* mixed_query,
                                     float* key_layer,
                                     unsigned rotary_dim,
                                     unsigned seq_len,
                                     unsigned seq_offset,
                                     unsigned num_heads,
                                     unsigned head_size,
                                     unsigned total_count,
                                     int max_out_tokens)
{
    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    int id = threadIdx.x;
    int gid = id >> 5;
    int lane = id & 0x1f;

    unsigned head_id = blockIdx.x * MAX_WARP_NUM + gid;
    unsigned offset = head_id * head_size;

    unsigned seq_id = (head_id / num_heads) % seq_len + seq_offset;
    unsigned seq_index = head_id % seq_len;
    unsigned k_offset = (seq_index + (head_id / seq_len) * max_out_tokens) * head_size;

    if (head_id < total_count) {
        while (lane < rotary_dim) {
            float inv_freq = (float)((lane / 2) * 2) / (float)rotary_dim;
            inv_freq = 1.0 / powf(10000.0, inv_freq) * (float)seq_id;
            float q = mixed_query[offset + lane];
            float k = key_layer[k_offset + lane];
            float rotary_sign = (lane % 2 == 1 ? -1.0 : 1.0);
            float q_rot = (q * rotary_sign);
            float k_rot = (k * rotary_sign);
            q_rot = g.shfl_xor(q_rot, 1);
            k_rot = g.shfl_xor(k_rot, 1);
            q = q * cosf(inv_freq) + q_rot * sinf(inv_freq);
            k = k * cosf(inv_freq) + k_rot * sinf(inv_freq);

            mixed_query[offset + lane] = q;
            key_layer[k_offset + lane] = k;

            lane += WARP_SIZE;
        }
    }
}

__global__ void apply_rotary_pos_emb(__half* mixed_query,
                                     __half* key_layer,
                                     unsigned rotary_dim,
                                     unsigned seq_len,
                                     unsigned seq_offset,
                                     unsigned num_heads,
                                     unsigned head_size,
                                     unsigned total_count,
                                     int max_out_tokens)
{
    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    int id = threadIdx.x;
    int gid = id >> 5;
    int lane = id & 0x1f;

    unsigned head_id = blockIdx.x * MAX_WARP_NUM + gid;
    unsigned offset = head_id * head_size;

    unsigned seq_id = (head_id / num_heads) % seq_len + seq_offset;
    unsigned seq_index = head_id % seq_len;
    unsigned k_offset = (seq_index + (head_id / seq_len) * max_out_tokens) * head_size;

    if (head_id < total_count) {
        while (lane < rotary_dim) {
            float inv_freq = (float)((lane / 2) * 2) / (float)rotary_dim;
            inv_freq = 1.0 / powf(10000.0, inv_freq) * (float)seq_id;
            float q = (float)mixed_query[offset + lane];
            float k = (float)key_layer[k_offset + lane];
            float rotary_sign = (lane % 2 == 1 ? -1.0 : 1.0);
            float q_rot = (q * rotary_sign);
            float k_rot = (k * rotary_sign);
            q_rot = g.shfl_xor(q_rot, 1);
            k_rot = g.shfl_xor(k_rot, 1);
            q = q * cosf(inv_freq) + q_rot * sinf(inv_freq);
            k = k * cosf(inv_freq) + k_rot * sinf(inv_freq);

            mixed_query[offset + lane] = (__half)q;
            key_layer[k_offset + lane] = (__half)k;

            lane += WARP_SIZE;
        }
    }
}
__global__ void apply_rotary_pos_emb1(float* mixed_query,
                                      float* key_layer,
                                      unsigned rotary_dim,
                                      unsigned seq_len,
                                      unsigned seq_offset,
                                      unsigned num_heads,
                                      unsigned head_size,
                                      unsigned total_count,
                                      int max_out_tokens)
{
    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    int id = threadIdx.x;
    int gid = id >> 5;
    int lane = id & 0x1f;

    unsigned head_id = blockIdx.x * MAX_WARP_NUM + gid;
    unsigned offset = head_id * head_size;

    unsigned seq_id = (head_id / num_heads) % seq_len + seq_offset;
    unsigned seq_index = head_id % seq_len;
    unsigned k_offset = (seq_index + (head_id / seq_len) * max_out_tokens) * head_size;

    if (head_id < total_count) {
        while (lane < rotary_dim) {
            float inv_freq = (float)((lane / 2) * 2) / (float)rotary_dim;
            inv_freq = 1.0 / powf(10000.0, inv_freq) * (float)seq_id;
            float q = mixed_query[offset + lane];
            float k = key_layer[k_offset + lane];
            float rotary_sign = (lane % 2 == 1 ? -1.0 : 1.0);
            float q_rot = (q * rotary_sign);
            float k_rot = (k * rotary_sign);
            q_rot = g.shfl_xor(q_rot, 1);
            k_rot = g.shfl_xor(k_rot, 1);
            q = q * cosf(inv_freq) + q_rot * sinf(inv_freq);
            k = k * cosf(inv_freq) + k_rot * sinf(inv_freq);

            mixed_query[offset + lane] = q;
            key_layer[k_offset + lane] = k;

            lane += WARP_SIZE;
        }
    }
}
__global__ void apply_rotary_pos_emb1(__half* mixed_query,
                                      __half* key_layer,
                                      unsigned rotary_dim,
                                      unsigned seq_len,
                                      unsigned seq_offset,
                                      unsigned num_heads,
                                      unsigned head_size,
                                      unsigned total_count,
                                      int max_out_tokens)
{
    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    int id = threadIdx.x;
    int gid = id >> 5;
    int lane = id & 0x1f;

    unsigned head_id = blockIdx.x * MAX_WARP_NUM + gid;
    unsigned seq_index = head_id % seq_len;
    unsigned offset = head_id * head_size;
    unsigned k_offset = (seq_index + (head_id / seq_len) * max_out_tokens) * head_size;

    constexpr unsigned mask[32] = {
        0x1 | 0x1000,     0x2 | 0x2000,     0x4 | 0x4000,     0x8 | 0x8000,     0x10 | 0x10000,
        0x20 | 0x20000,   0x40 | 0x40000,   0x80 | 0x80000,   0x100 | 0x100000, 0x200 | 0x200000,
        0x400 | 0x400000, 0x800 | 0x800000, 0x1000 | 0x1,     0x2000 | 0x2,     0x4000 | 0x4,
        0x8000 | 0x8,     0x10000 | 0x10,   0x20000 | 0x20,   0x40000 | 0x40,   0x80000 | 0x80,
        0x100000 | 0x100, 0x200000 | 0x200, 0x400000 | 0x400, 0x800000 | 0x800, 0x1000000,
        0x2000000,        0x4000000,        0x8000000,        0x10000000,       0x20000000,
        0x40000000,       0x80000000};

    unsigned seq_id = (head_id % seq_len) + seq_offset;
    unsigned half_dim = rotary_dim >> 1;
    if (head_id < total_count) {
        while (lane < rotary_dim) {
            float inv_freq = (float)((lane % half_dim) * 2) / (float)rotary_dim;
            inv_freq = 1.0 / powf(10000.0, inv_freq) * (float)seq_id;
            float q = (float)mixed_query[offset + lane];
            float k = (float)key_layer[k_offset + lane];
            float rotary_sign = (lane > (half_dim - 1) ? -1.0 : 1.0);
            float q_rot = (q * rotary_sign);
            float k_rot = (k * rotary_sign);
            auto q_rot_tmp = lane < half_dim ? __shfl_sync(mask[lane], q_rot, lane + half_dim)
                                             : __shfl_sync(mask[lane], q_rot, lane - half_dim);
            auto k_rot_tmp = lane < half_dim ? __shfl_sync(mask[lane], k_rot, lane + half_dim)
                                             : __shfl_sync(mask[lane], k_rot, lane - half_dim);
            q = q * cosf(inv_freq) + q_rot_tmp * sinf(inv_freq);
            k = k * cosf(inv_freq) + k_rot_tmp * sinf(inv_freq);

            mixed_query[offset + lane] = (__half)q;
            key_layer[k_offset + lane] = (__half)k;

            lane += WARP_SIZE;
        }
    }
}

template <typename T>
void launch_apply_rotary_pos_emb(T* mixed_query,
                                 T* key_layer,
                                 unsigned head_size,
                                 unsigned seq_len,
                                 unsigned rotary_dim,
                                 unsigned offset,
                                 unsigned num_heads,
                                 unsigned batch,
                                 bool rotate_half,
                                 bool rotate_every_two,
                                 hipStream_t stream,
                                 int max_out_tokens)
{
    int total_count = batch * num_heads * seq_len;
    dim3 block_dims(1024);
    dim3 grid_dims((total_count - 1) / MAX_WARP_NUM + 1);  // (batch_size);
    if (rotate_every_two)
        apply_rotary_pos_emb<<<grid_dims, block_dims, 0, stream>>>(mixed_query,
                                                                   key_layer,
                                                                   rotary_dim,
                                                                   seq_len,
                                                                   offset,
                                                                   num_heads,
                                                                   head_size,
                                                                   total_count,
                                                                   max_out_tokens);
    else if (rotate_half)
        apply_rotary_pos_emb1<<<grid_dims, block_dims, 0, stream>>>(mixed_query,
                                                                    key_layer,
                                                                    rotary_dim,
                                                                    seq_len,
                                                                    offset,
                                                                    num_heads,
                                                                    head_size,
                                                                    total_count,
                                                                    max_out_tokens);
}

template void launch_apply_rotary_pos_emb<float>(float*,
                                                 float*,
                                                 unsigned,
                                                 unsigned,
                                                 unsigned,
                                                 unsigned,
                                                 unsigned,
                                                 unsigned,
                                                 bool,
                                                 bool,
                                                 hipStream_t,
                                                 int);
template void launch_apply_rotary_pos_emb<__half>(__half*,
                                                  __half*,
                                                  unsigned,
                                                  unsigned,
                                                  unsigned,
                                                  unsigned,
                                                  unsigned,
                                                  unsigned,
                                                  bool,
                                                  bool,
                                                  hipStream_t,
                                                  int);

/*
__global__ void apply_rotary_pos_emb(float* mixed_query,
float* key_layer,
unsigned rotary_dim,
unsigned seq_len,
unsigned seq_offset,
unsigned num_heads,
unsigned head_size,
unsigned total_count)
{
cg::thread_block b = cg::this_thread_block();
cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

int id = threadIdx.x;
int gid = id >> 5;
int lane = id & 0x1f;

unsigned head_id = blockIdx.x * MAX_WARP_NUM + gid;
unsigned offset = head_id * head_size;

unsigned seq_id = (head_id / num_heads) % seq_len + seq_offset;

if (head_id < total_count) {
while (lane < rotary_dim) {
float inv_freq = (float)((lane / 2) * 2) / (float)rotary_dim;
inv_freq = 1.0 / powf(10000.0, inv_freq) * (float)seq_id;
float q = mixed_query[offset + lane];
float k = key_layer[offset + lane];
float rotary_sign = (lane % 2 == 1 ? -1.0 : 1.0);
float q_rot = (q * rotary_sign);
float k_rot = (k * rotary_sign);
q_rot = g.shfl_xor(q_rot, 1);
k_rot = g.shfl_xor(k_rot, 1);
q = q * cosf(inv_freq) + q_rot * sinf(inv_freq);
k = k * cosf(inv_freq) + k_rot * sinf(inv_freq);

mixed_query[offset + lane] = q;
key_layer[offset + lane] = k;

lane += WARP_SIZE;
}
}
}

__global__ void apply_rotary_pos_emb(__half* mixed_query,
__half* key_layer,
unsigned rotary_dim,
unsigned seq_len,
unsigned seq_offset,
unsigned num_heads,
unsigned head_size,
unsigned total_count)
{
#if __CUDA_ARCH__ >= 700
cg::thread_block b = cg::this_thread_block();
cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

int id = threadIdx.x;
int gid = id >> 5;
int lane = id & 0x1f;

unsigned head_id = blockIdx.x * MAX_WARP_NUM + gid;
unsigned offset = head_id * head_size;
constexpr unsigned mask[32] = {0x1 | 0x1000, 0x2 | 0x2000, 0x4 | 0x4000, 0x8 | 0x8000,
0x10 | 0x10000, 0x20 | 0x20000, 0x40 | 0x40000, 0x80 | 0x80000,
0x100 | 0x100000, 0x200 | 0x200000, 0x400 | 0x400000, 0x800 | 0x800000,
0x1000 | 0x1, 0x2000 | 0x2, 0x4000 | 0x4, 0x8000 | 0x8,
0x10000 | 0x10, 0x20000 | 0x20, 0x40000 | 0x40, 0x80000 | 0x80,
0x100000 | 0x100, 0x200000 | 0x200, 0x400000 | 0x400, 0x800000 | 0x800,
0x1000000, 0x2000000, 0x4000000, 0x8000000,
0x10000000, 0x20000000, 0x40000000, 0x80000000};
unsigned seq_id = (head_id / num_heads) % seq_len + seq_offset;

if (head_id < total_count) {
while (lane < rotary_dim) {
//float inv_freq = (float)((lane / 2) * 2) / (float)rotary_dim;
float inv_freq = (float)((lane % (rotary_dim >> 1)) * 2) / (float)rotary_dim;
inv_freq = 1.0 / powf(10000.0, inv_freq) * (float)seq_id;
float q = (float)mixed_query[offset + lane];
float k = (float)key_layer[offset + lane];
float rotary_sign = (lane > 11 ? -1.0 : 1.0);
float q_rot = (q * rotary_sign);
float k_rot = (k * rotary_sign);
auto q_rot_tmp = lane < 12 ? __shfl_sync(mask[lane], q_rot, lane + 12) : __shfl_sync(mask[lane],
q_rot, lane - 12);//g.shfl_xor(q_rot, 12); auto k_rot_tmp = lane < 12 ? __shfl_sync(mask[lane],
k_rot, lane + 12) : __shfl_sync(mask[lane], k_rot, lane - 12);//g.shfl_xor(k_rot, 12); q = q *
cosf(inv_freq) + q_rot_tmp * sinf(inv_freq); k = k * cosf(inv_freq) + k_rot_tmp * sinf(inv_freq);

mixed_query[offset + lane] = (__half)q;
key_layer[offset + lane] = (__half)k;

lane += WARP_SIZE;
}
}
#endif
}

template <typename T>
void launch_apply_rotary_pos_emb(T* mixed_query,
T* key_layer,
unsigned head_size,
unsigned seq_len,
unsigned rotary_dim,
unsigned offset,
unsigned num_heads,
unsigned batch,
hipStream_t stream)
{
int total_count = batch * num_heads * seq_len;
dim3 block_dims(1024);
dim3 grid_dims((total_count - 1) / MAX_WARP_NUM + 1);  // (batch_size);

apply_rotary_pos_emb<<<grid_dims, block_dims, 0, stream>>>(
mixed_query, key_layer, rotary_dim, seq_len, offset, num_heads, head_size, total_count);
}

template void launch_apply_rotary_pos_emb<float>(float*,
float*,
unsigned,
unsigned,
unsigned,
unsigned,
unsigned,
unsigned,
hipStream_t);
template void launch_apply_rotary_pos_emb<__half>(__half*,
__half*,
unsigned,
unsigned,
unsigned,
unsigned,
unsigned,
unsigned,
hipStream_t);
*/
